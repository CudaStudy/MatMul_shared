#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define M 1024
#define N 2048
#define K 1024

#define A_SIZE (M*K)
#define B_SIZE (K*N)
#define C_SIZE (M*N)

#define BLOCKSIZE 32

__device__ int getIdx(int rowIdx, int colIdx, int colSize){
    return rowIdx * colSize + colIdx;
}

/* KERNEL FUNCTION */
__global__ void matMul(float *_A, float *_B, float *_C){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
    int localrow = threadIdx.y;
    int localcol = threadIdx.x;
    float val = 0.0;

    __shared__ float sA[BLOCKSIZE][BLOCKSIZE];
    __shared__ float sB[BLOCKSIZE][BLOCKSIZE];

    for (int iter = 0; iter < K/BLOCKSIZE; iter++){
        // Copy to shared memory
        sA[localrow][localcol] = _A[getIdx(row, iter * BLOCKSIZE + localcol, K)];
        sB[localrow][localcol] = _B[getIdx(iter * BLOCKSIZE + localrow, col, N)];
        __syncthreads();

        // Compute
        for (int i = 0; i < BLOCKSIZE; i++){
            val += sA[localrow][i] * sB[i][localcol];
        }
        __syncthreads();
    }

    // Write results
    _C[getIdx(row, col, N)] = val;
}

/* MAIN FUNCTION */
int main(void){
    /*
    float A[M][K];
    float B[K][N];
    float hostC[M][N];
    float deviceC[M][N];
    */

    float *dA, *dB, *dC;
    dA = dB = dC = NULL;

    dim3 dimGrid(ceil((float)N / BLOCKSIZE), ceil((float)M / BLOCKSIZE), 1);
    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);

    struct timeval startTime, endTime;
    double elapsedTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time = 0;

    int numThreads = BLOCKSIZE * BLOCKSIZE;
    int numBlocks = ceil((float)M / BLOCKSIZE) * ceil((float)N / BLOCKSIZE);
    int numData = C_SIZE;
    int numOps = numData > (numBlocks * numThreads) ? (numData / (numBlocks * numThreads)) + (numData % (numBlocks * numThreads) > 0 ? 1 : 0): 1;
    printf("Need %d threads\n", numData);
    printf("numOps : %d\n", numOps);

    // Host memory allocation
    float* A = new float[A_SIZE];
    float* B = new float[B_SIZE];
    float* hostC = new float[C_SIZE];
    float* deviceC = new float[C_SIZE];

    for (int i = 0; i < M; i++){
        for (int j = 0; j < K; j++){
            A[i*K+j] = rand() % 100;
        }
    }
    for (int i = 0; i < K; i++){
        for (int j = 0; j < N; j++){
            B[i*N+j] = rand() % 100;
        }
    }   

    // Device memory allocation
    hipMalloc(&dA, sizeof(float)*A_SIZE);
    hipMalloc(&dB, sizeof(float)*B_SIZE);
    hipMalloc(&dC, sizeof(float)*C_SIZE);

    // Copy input data from host to device
    hipMemcpy(dA, A, sizeof(float)*A_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float)*B_SIZE, hipMemcpyHostToDevice);
    
    // Kernel call (GPU computation)
    hipEventRecord(start, 0);
    matMul<<<dimGrid, dimBlock>>>(dA, dB, dC);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy output data from device to host
    hipMemcpy(deviceC, dC, sizeof(float)*C_SIZE, hipMemcpyDeviceToHost);

    // CPU computation
    gettimeofday(&startTime, NULL);
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            hostC[i*N+j] = 0.0;
            for (int k = 0; k < K; k++){
                hostC[i*N+j] += A[i*K+k] * B[k*N+j];
            }
        }
    }
    gettimeofday(&endTime, NULL);
    elapsedTime += (endTime.tv_sec - startTime.tv_sec) * 1000. + (endTime.tv_usec - startTime.tv_usec) / 1000.;  // in ms

    // Check results
    bool result = true;
    for (int i = 0; i < M; i++){
        for (int j = 0; j < N; j++){
            if (hostC[i*N+j] != deviceC[i*N+j]){
                printf("[%d]th result is not matched! (CPU : %f vs GPU : %f)\n", i*N+j, hostC[i*N+j], deviceC[i*N+j]);
                result = false;                
            }
        }
    }

    if (result){
        printf("GPU works well-!\n");
    }

    printf("-------------------------------------------------\n");
    printf("# of blocks : %d, # of threads : %d\n", numBlocks, numThreads);
    printf("-------------------------------------------------\n");
    printf("CPU execution time : %fms\n", elapsedTime);
    printf("GPU execution time : %fms\n", time);
    printf("-------------------------------------------------\n");
    printf("GPU speedup over CPU : %f\n", elapsedTime / time);

    // Host & Device memory release
    delete [] A;
    delete [] B;
    delete [] hostC;
    delete [] deviceC;
 
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
